#include "hip/hip_runtime.h"
#include "mlx/backend/common/matmul.h"
#include "mlx/backend/cuda/device.h"
#include "mlx/backend/cuda/device/utils.cuh"
#include "mlx/backend/cuda/gemms/steel_gemm.h"
#include "mlx/backend/cuda/kernel_utils.cuh"
#include "mlx/primitives.h"

#include <nvtx3/nvtx3.hpp>
#include <numeric>

#include <hip/hip_cooperative_groups.h>

#include "mlx/backend/cuda/steel/gemm.cuh"
#include "mlx/backend/cuda/steel/mma.cuh"
#include "mlx/backend/cuda/steel/tiles.cuh"

namespace mlx::core {

namespace cu {

namespace cg = cooperative_groups;

struct GemmParams {
  int M;
  int N;
  int K;
  int lda;
  int ldb;
  int ldd;

  int NblockM;
  int NblockN;
  int NblockK;
};

template <
    typename T,
    int BM,
    int BN,
    int BK,
    int WM,
    int WN,
    bool transpose_a,
    bool transpose_b,
    int SL,
    int Nstages>
__global__ void kernel_steel_gemm(
    const T* a,
    const T* b,
    T* d,
    __grid_constant__ const GemmParams params) {
  const int bM_idx = (blockIdx.y << SL) + (blockIdx.x & ((1 << SL) - 1));
  const int bN_idx = blockIdx.x >> SL;

  if (params.NblockN <= bN_idx || params.NblockM <= bM_idx) {
    return;
  }

  const int d_row = bM_idx * BM;
  const int d_col = bN_idx * BN;
  const size_t d_row_long = size_t(d_row);
  const size_t d_col_long = size_t(d_col);

  a += transpose_a ? d_row_long : d_row_long * params.K;
  b += transpose_b ? d_col_long * params.K : d_col_long;
  d += d_row_long * params.ldd + d_col_long;

  auto block = cg::this_thread_block();
  auto warp = cg::tiled_partition<32>(block);

  const int lane_idx = warp.thread_rank();
  const int warp_idx = warp.meta_group_rank();

  const int wm = warp_idx / WN;
  const int wn = warp_idx % WN;

  constexpr int SM = BM / WM;
  constexpr int SN = BN / WN;
  constexpr int SK = BK;
  constexpr int TK = SK / 16;

  constexpr int NUM_WARPS = WM * WN;

  // Allocate shared memory
  extern __shared__ char shmem[];
  SharedTile<T, BM, BK>(&as)[Nstages] =
      *(SharedTile<T, BM, BK>(*)[Nstages])(&shmem[0]);
  SharedTile<T, BN, BK>(&bs)[Nstages] = *(SharedTile<T, BN, BK>(*)[Nstages])(
      &shmem[sizeof(T) * Nstages * BM * BK]);

  // Allocate registers for the MMA
  RegisterTile<float, SM, SN> C;
  RegisterTile<T, SM, 16> A[TK];
  RegisterTile<T, SN, 16> B[TK];

  // Zero the accumulators
  C.fill(0);

  // Start gmem -> smem copies
  int k_block_read = 0;

  MLX_UNROLL
  for (int bk = 0; bk < (Nstages - 1); bk++) {
    load_async<NUM_WARPS>(
        as[bk], as[bk].base_addr(), a + k_block_read, params.K);
    load_async<NUM_WARPS>(
        bs[bk], bs[bk].base_addr(), b + k_block_read, params.K);
    k_block_read += BK;
    cp_async_commit();
  }

  int smem_pipe_read = 0;
  int smem_pipe_write = Nstages - 1;

  // Wait till only 1 remains laoding
  cp_async_wait<1>();
  block.sync();

  const int offset_m = wm * SM;
  const int offset_n = wn * SN;

  // Start smem -> register copy
  A[0].load(
      as[smem_pipe_read],
      as[smem_pipe_read].base_addr(),
      offset_m + lane_idx % 16,
      lane_idx / 16 * 8);
  B[0].load(
      bs[smem_pipe_read],
      bs[smem_pipe_read].base_addr(),
      offset_n + lane_idx % 16,
      lane_idx / 16 * 8);

  // Main loop
  for (int kb = 0; kb < params.NblockK; kb++) {
    // Prepare next registers
    {
      A[1].load(
          as[smem_pipe_read],
          as[smem_pipe_read].base_addr(),
          offset_m + lane_idx % 16,
          16 + lane_idx / 16 * 8);
      B[1].load(
          bs[smem_pipe_read],
          bs[smem_pipe_read].base_addr(),
          offset_n + lane_idx % 16,
          16 + lane_idx / 16 * 8);
    }

    // Prepare next smem
    if ((kb + Nstages - 1) < params.NblockK) {
      load_async<NUM_WARPS>(
          as[smem_pipe_write],
          as[smem_pipe_write].base_addr(),
          a + k_block_read,
          params.K);
      load_async<NUM_WARPS>(
          bs[smem_pipe_write],
          bs[smem_pipe_write].base_addr(),
          b + k_block_read,
          params.K);
    }
    k_block_read += BK;

    cp_async_commit();

    smem_pipe_write = smem_pipe_read;
    smem_pipe_read = smem_pipe_read + 1;
    smem_pipe_read = (smem_pipe_read == Nstages) ? 0 : smem_pipe_read;

    // Do current gemm
    mma_t(C, A[0], B[0]);

    // Do wait for next register
    cp_async_wait<1>();
    block.sync();

    // Prepare next register (smem_pipe_read has moved to the next)
    {
      A[0].load(
          as[smem_pipe_read],
          as[smem_pipe_read].base_addr(),
          offset_m + lane_idx % 16,
          lane_idx / 16 * 8);
      B[0].load(
          bs[smem_pipe_read],
          bs[smem_pipe_read].base_addr(),
          offset_n + lane_idx % 16,
          lane_idx / 16 * 8);
    }

    // Do current gemm
    mma_t(C, A[1], B[1]);
  }

  // Wait and clear
  cp_async_wait_all();
  block.sync();

  C.store_global(d, params.ldd, offset_m, offset_n);
}

} // namespace cu

void dispatch_steel_gemm(
    const Stream& s,
    cu::CommandEncoder& encoder,
    const array& a,
    const array& b,
    array& d,
    int M,
    int N,
    int K,
    int lda,
    int ldb,
    int ldd,
    bool a_transposed,
    bool b_transposed) {
  using DataType = cuda_type_t<float16_t>;

  encoder.set_input_array(a);
  encoder.set_input_array(b);
  encoder.set_output_array(d);

  constexpr int BM = 128;
  constexpr int BN = 128;
  constexpr int BK = 32;

  constexpr int WM = 2;
  constexpr int WN = 2;

  constexpr int SL = 0;
  constexpr int Nstages = 3;

  constexpr uint32_t smem_bytes = BK * (BM + BN) * Nstages * sizeof(DataType);

  const int NblockM = (M + BM - 1) / BM;
  const int NblockN = (N + BN - 1) / BN;
  const int NblockK = (K + BK - 1) / BK;

  cu::GemmParams params{
      /* int M = */ M,
      /* int N = */ N,
      /* int K = */ K,
      /* int lda = */ lda,
      /* int ldb = */ ldb,
      /* int ldd = */ ldd,

      /* int NblockM = */ NblockM,
      /* int NblockN = */ NblockN,
      /* int NblockK = */ NblockK,
  };

  // Prepare launch grid params
  int tile = 1 << SL;
  int tm = (NblockM + tile - 1) / tile;
  int tn = NblockN * tile;

  dim3 grid_dim(tn, tm, 1);
  dim3 block_dim(32 * WM * WN, 1, 1);

  dispatch_bool(a_transposed, [&](auto ta_) {
    dispatch_bool(b_transposed, [&](auto tb_) {
      constexpr bool ta = ta_.value;
      constexpr bool tb = tb_.value;

      auto kernel = cu::ab_t_aligned<DataType, BM, BN, BK>;
      hipFuncSetAttribute(reinterpret_cast<const void*>(
          kernel), hipFuncAttributeMaxDynamicSharedMemorySize, smem_bytes);

      encoder.add_kernel_node(
          kernel,
          grid_dim,
          block_dim,
          smem_bytes,
          a.data<DataType>(),
          b.data<DataType>(),
          d.data<DataType>(),
          N,
          K);

      //   auto kernel = cu::kernel_steel_gemm<DataType, BM, BN, BK, WM, WN, ta,
      //   tb, SL, Nstages>;

      //   hipFuncSetAttribute(reinterpret_cast<const void*>(kernel),
      //   hipFuncAttributeMaxDynamicSharedMemorySize, smem_bytes);

      //   encoder.add_kernel_node(
      //       kernel,
      //       grid_dim,
      //       block_dim,
      //       smem_bytes,
      //       a.data<DataType>(),
      //       b.data<DataType>(),
      //       d.data<DataType>(),
      //       params);
    });
  });
}

} // namespace mlx::core