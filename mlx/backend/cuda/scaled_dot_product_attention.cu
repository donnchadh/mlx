#include "hip/hip_runtime.h"
// Copyright © 2025 Apple Inc.

#include "mlx/backend/cuda/device.h"
#include "mlx/backend/cuda/device/config.h"
#include "mlx/backend/cuda/device/utils.cuh"
#include "mlx/backend/cuda/kernel_utils.cuh"
#include "mlx/backend/cuda/lru_cache.h"
#include "mlx/backend/gpu/copy.h"
#include "mlx/dtype_utils.h"
#include "mlx/fast_primitives.h"

// cudnn_frontend.h redefines this macro.
#undef CHECK_CUDA_ERROR

#include <cudnn_frontend.h>
#include <fmt/format.h>
#include <nvtx3/nvtx3.hpp>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

namespace fe = cudnn_frontend;

namespace mlx::core {

namespace cu {

namespace cg = cooperative_groups;

#define PRAGMA_LOOP_UNROLL #pragma unroll

struct AttnParams {
  int B;
  int H;
  int D;

  int qL;
  int kL;

  int gqa_factor;
  float scale;

  int64_t Q_strides[3];
  int64_t K_strides[3];
  int64_t V_strides[3];
  int64_t O_strides[3];
};

template <typename T, bool do_causal, int D>
__global__ void kernel_sdpav_1pass(
    const T* Q,
    const T* K,
    const T* V,
    T* O,
    __grid_constant__ const AttnParams params) {
  constexpr int BN = 32;
  constexpr int BD = 32;

  constexpr int v_per_thread = D / BD;

  const int inner_k_stride = BN * int(params.K_strides[2]);
  const int inner_v_stride = BN * int(params.V_strides[2]);

  typedef float U;

  U q[v_per_thread];
  U k[v_per_thread];
  U o[v_per_thread];

  __shared__ U outputs[BN][BD + 1];
  __shared__ U max_scores[BN];
  __shared__ U sum_exp_scores[BN];

  const U scale_log2 = params.scale * 1.44269504089f;

  auto block = cg::this_thread_block();
  auto warp = cg::tiled_partition<32>(block);

  const int lane_idx = warp.thread_rank();
  const int warp_idx = warp.meta_group_rank();

  // Adjust to thread block and thread
  const int batch_idx = blockIdx.z;
  const int head_idx = blockIdx.x;
  const int kv_head_idx = head_idx / params.gqa_factor;

  const int q_seq_idx = blockIdx.y;
  const int kv_seq_idx = warp_idx;

  Q += batch_idx * params.Q_strides[0] + // Batch
      head_idx * params.Q_strides[1] + // Head
      q_seq_idx * params.Q_strides[2]; // Sequence

  K += batch_idx * params.K_strides[0] + // Batch
      kv_head_idx * params.K_strides[1] + // Head
      kv_seq_idx * params.K_strides[2]; // Sequence

  V += batch_idx * params.V_strides[0] + // Batch
      kv_head_idx * params.V_strides[1] + // Head
      kv_seq_idx * params.V_strides[2]; // Sequence

  O += batch_idx * params.O_strides[0] + // Batch
      head_idx * params.O_strides[1] + // Head
      q_seq_idx * params.O_strides[2]; // Sequence

  // Read the query and 0 the output accumulator
  PRAGMA_LOOP_UNROLL
  for (int i = 0; i < v_per_thread; i++) {
    q[i] = scale_log2 * static_cast<U>(Q[v_per_thread * lane_idx + i]);
  }

  PRAGMA_LOOP_UNROLL
  for (int i = 0; i < v_per_thread; i++) {
    o[i] = 0.f;
  }

  U max_score = -INFINITY;
  U sum_exp_score = 0.f;

  // For each key
  for (int i = kv_seq_idx; i < params.kL; i += BN) {
    bool use_key = true;
    if constexpr (do_causal) {
      use_key = i <= (params.kL - params.qL + q_seq_idx);
    }

    if (use_key) {
      // Read the key
      PRAGMA_LOOP_UNROLL
      for (int j = 0; j < v_per_thread; j++) {
        k[j] = K[v_per_thread * lane_idx + j];
      }

      // Compute the i-th score
      U score = 0.f;
      PRAGMA_LOOP_UNROLL
      for (int j = 0; j < v_per_thread; j++) {
        score += q[j] * k[j];
      }

      // Warp sum
      score = cg::reduce(warp, score, cg::plus<U>());

      // Update the accumulators
      U new_max = max(max_score, score);
      U factor = exp2f(max_score - new_max);
      U exp_score = exp2f(score - new_max);

      max_score = new_max;
      sum_exp_score = sum_exp_score * factor + exp_score;

      // Update the output accumulator
      PRAGMA_LOOP_UNROLL
      for (int j = 0; j < v_per_thread; j++) {
        o[j] = o[j] * factor +
            exp_score * static_cast<U>(V[v_per_thread * lane_idx + j]);
      }
    }

    // Move the pointers to the next kv
    K += inner_k_stride;
    V += inner_v_stride;
  }

  if (lane_idx == 0) {
    max_scores[warp_idx] = max_score;
    sum_exp_scores[warp_idx] = sum_exp_score;
  }
  block.sync();

  max_score = max_scores[lane_idx];
  U new_max = cg::reduce(warp, max_score, cg::greater<U>());
  U factor = exp2f(max_score - new_max);
  sum_exp_score =
      cg::reduce(warp, sum_exp_scores[lane_idx] * factor, cg::plus<U>());
  sum_exp_score = __frcp_rn(sum_exp_score);

  // Now we need to aggregate all the outputs
  PRAGMA_LOOP_UNROLL
  for (int i = 0; i < v_per_thread; i++) {
    outputs[lane_idx][warp_idx] = o[i];
    block.sync();
    U ot = outputs[warp_idx][lane_idx] * factor;
    o[i] = cg::reduce(warp, ot, cg::plus<U>()) * sum_exp_score;
    block.sync();
  }

  // And write the output
  if (lane_idx == 0) {
    PRAGMA_LOOP_UNROLL
    for (int i = 0; i < v_per_thread; i++) {
      O[v_per_thread * warp_idx + i] = static_cast<T>(o[i]);
    }
  }
}

template <typename T, bool do_causal, int D>
__global__ void kernel_sdpav_2pass_1(
    const T* Q,
    const T* K,
    const T* V,
    float* partials,
    float* sums,
    float* maxs,
    __grid_constant__ const AttnParams params) {
  constexpr int BN = 8;
  constexpr int BD = 32;
  constexpr int blocks = 32;

  constexpr int v_per_thread = D / BD;

  const int inner_k_stride = blocks * BN * int(params.K_strides[2]);
  const int inner_v_stride = blocks * BN * int(params.V_strides[2]);

  typedef float U;

  U q[v_per_thread];
  U k[v_per_thread];
  U o[v_per_thread];

  __shared__ U outputs[BN][BD + 1];
  __shared__ U max_scores[BN];
  __shared__ U sum_exp_scores[BN];

  const U scale_log2 = params.scale * 1.44269504089f;

  auto block = cg::this_thread_block();
  auto warp = cg::tiled_partition<32>(block);

  const int lane_idx = warp.thread_rank();
  const int warp_idx = warp.meta_group_rank();

  // Adjust to thread block and thread
  const int batch_idx = blockIdx.z / blocks;
  const int block_idx = blockIdx.z % blocks;
  const int head_idx = blockIdx.x;
  const int kv_head_idx = head_idx / params.gqa_factor;

  const int q_seq_idx = blockIdx.y;
  const int kv_seq_idx = block_idx * BN + warp_idx;

  Q += batch_idx * params.Q_strides[0] + // Batch
      head_idx * params.Q_strides[1] + // Head
      q_seq_idx * params.Q_strides[2]; // Sequence

  K += batch_idx * params.K_strides[0] + // Batch
      kv_head_idx * params.K_strides[1] + // Head
      kv_seq_idx * params.K_strides[2]; // Sequence

  V += batch_idx * params.V_strides[0] + // Batch
      kv_head_idx * params.V_strides[1] + // Head
      kv_seq_idx * params.V_strides[2]; // Sequence

  const int p_stride_s = blocks;
  const int p_stride_h = params.qL * p_stride_s;
  const int p_stride_b = params.H * p_stride_h;
  const int p_offset = batch_idx * p_stride_b + // Batch
      head_idx * p_stride_h + // Head
      q_seq_idx * p_stride_s + // Sequence
      block_idx; // Block

  partials += p_offset * D;
  sums += p_offset;
  maxs += p_offset;

  // Read the query and 0 the output accumulator
  PRAGMA_LOOP_UNROLL
  for (int i = 0; i < v_per_thread; i++) {
    q[i] = scale_log2 * static_cast<U>(Q[v_per_thread * lane_idx + i]);
  }

  PRAGMA_LOOP_UNROLL
  for (int i = 0; i < v_per_thread; i++) {
    o[i] = 0.f;
  }

  U max_score = -1e9;
  U sum_exp_score = 0.f;

  // For each key
  for (int i = kv_seq_idx; i < params.kL; i += blocks * BN) {
    bool use_key = true;
    if constexpr (do_causal) {
      use_key = i <= (params.kL - params.qL + q_seq_idx);
    }

    if (use_key) {
      // Read the key
      PRAGMA_LOOP_UNROLL
      for (int j = 0; j < v_per_thread; j++) {
        k[j] = K[v_per_thread * lane_idx + j];
      }

      // Compute the i-th score
      U score = 0.f;
      PRAGMA_LOOP_UNROLL
      for (int j = 0; j < v_per_thread; j++) {
        score += q[j] * k[j];
      }

      // Warp sum
      score = cg::reduce(warp, score, cg::plus<U>());

      // Update the accumulators
      U new_max = max(max_score, score);
      U factor = exp2f(max_score - new_max);
      U exp_score = exp2f(score - new_max);

      max_score = new_max;
      sum_exp_score = sum_exp_score * factor + exp_score;

      // Update the output accumulator
      PRAGMA_LOOP_UNROLL
      for (int j = 0; j < v_per_thread; j++) {
        o[j] = o[j] * factor +
            exp_score * static_cast<U>(V[v_per_thread * lane_idx + j]);
      }
    }

    // Move the pointers to the next kv
    K += inner_k_stride;
    V += inner_v_stride;
  }

  if (lane_idx == 0) {
    max_scores[warp_idx] = max_score;
    sum_exp_scores[warp_idx] = sum_exp_score;
  }

  block.sync();

  max_score = (lane_idx < BN) ? max_scores[lane_idx] : -1e9;
  U new_max = cg::reduce(warp, max_score, cg::greater<U>());
  U factor = exp2f(max_score - new_max);
  sum_exp_score = (lane_idx < BN) ? sum_exp_scores[lane_idx] : 0.f;
  sum_exp_score = cg::reduce(warp, sum_exp_score * factor, cg::plus<U>());

  // Write the sum and new max
  if (warp_idx == 0) {
    sums[0] = sum_exp_score;
    maxs[0] = new_max;
  }

  // Now we need to aggregate all the outputs
  auto ff = exp2f(max_scores[warp_idx] - new_max);
  PRAGMA_LOOP_UNROLL
  for (int i = 0; i < v_per_thread; i++) {
    outputs[warp_idx][lane_idx] = o[i] * ff;
    block.sync();

    if (warp_idx == 0) {
      U ot = outputs[0][lane_idx];
      PRAGMA_LOOP_UNROLL
      for (int j = 1; j < BN; j++) {
        ot += outputs[j][lane_idx];
        warp.sync();
      }
      o[i] = ot;
    }
    block.sync();
  }

  if (warp_idx == 0) {
    PRAGMA_LOOP_UNROLL
    for (int i = 0; i < v_per_thread; i++) {
      partials[v_per_thread * lane_idx + i] = o[i];
    }
  }
}

template <typename T, bool do_causal, int D>
__global__ void kernel_sdpav_2pass_2(
    const float* partials,
    const float* sums,
    const float* maxs,
    T* O,
    __grid_constant__ const AttnParams params) {
  constexpr int BN = 32;
  constexpr int BD = 32;
  constexpr int blocks = 32;

  constexpr int v_per_thread = D / BD;

  typedef float U;

  U o[v_per_thread];
  __shared__ U outputs[BN][BD + 1];

  auto block = cg::this_thread_block();
  auto warp = cg::tiled_partition<32>(block);

  const int lane_idx = warp.thread_rank();
  const int warp_idx = warp.meta_group_rank();

  // Adjust to thread block and thread
  const int batch_idx = blockIdx.z;
  const int head_idx = blockIdx.x;
  const int q_seq_idx = blockIdx.y;

  const int p_stride_s = blocks;
  const int p_stride_h = params.qL * p_stride_s;
  const int p_stride_b = params.H * p_stride_h;
  const int p_offset = batch_idx * p_stride_b + // Batch
      head_idx * p_stride_h + // Head
      q_seq_idx * p_stride_s; // Sequence

  partials += p_offset * D + warp_idx * D;
  sums += p_offset;
  maxs += p_offset;

  O += batch_idx * params.O_strides[0] + // Batch
      head_idx * params.O_strides[1] + // Head
      q_seq_idx * params.O_strides[2]; // Sequence

  U max_score = maxs[lane_idx];
  U new_max = cg::reduce(warp, max_score, cg::greater<U>());
  U factor = exp2f(max_score - new_max);
  U sum_exp_score = cg::reduce(warp, sums[lane_idx] * factor, cg::plus<U>());
  sum_exp_score = __frcp_rn(sum_exp_score);

  PRAGMA_LOOP_UNROLL
  for (int i = 0; i < v_per_thread; i++) {
    o[i] = partials[v_per_thread * lane_idx + i];
  }

  // Now we need to aggregate all the outputs
  PRAGMA_LOOP_UNROLL
  for (int i = 0; i < v_per_thread; i++) {
    outputs[lane_idx][warp_idx] = o[i];
    block.sync();
    U ot = outputs[warp_idx][lane_idx] * factor;
    o[i] = cg::reduce(warp, ot, cg::plus<U>()) * sum_exp_score;
    block.sync();
  }

  // And write the output
  if (lane_idx == 0) {
    PRAGMA_LOOP_UNROLL
    for (int i = 0; i < v_per_thread; i++) {
      O[v_per_thread * warp_idx + i] = static_cast<T>(o[i]);
    }
  }
}

} // namespace cu

namespace {

template <typename F>
void dispatch_headdim(int n, F&& f) {
  switch (n) {
    case 64:
      f(std::integral_constant<int, 64>{});
      break;
    case 96:
      f(std::integral_constant<int, 96>{});
      break;
    case 128:
      f(std::integral_constant<int, 128>{});
      break;
  }
}

void sdpa_vector_1pass_fallback(
    const Stream& s,
    cu::CommandEncoder& encoder,
    const array& q,
    const array& k,
    const array& v,
    const float scale,
    array& o,
    bool do_causal_ = false) {
  encoder.set_input_array(q);
  encoder.set_input_array(k);
  encoder.set_input_array(v);
  encoder.set_output_array(o);

  cu::AttnParams params{
      /* int B = */ q.shape(0),
      /* int H = */ q.shape(1),
      /* int D = */ q.shape(3),

      /* int qL = */ q.shape(2),
      /* int kL = */ k.shape(2),

      /* int gqa_factor = */ q.shape(1) / k.shape(1),
      /* float scale = */ scale,

      /* int64_t Q_strides[3] = */ {q.strides(0), q.strides(1), q.strides(2)},
      /* int64_t K_strides[3] = */ {k.strides(0), k.strides(1), k.strides(2)},
      /* int64_t V_strides[3] = */ {v.strides(0), v.strides(1), v.strides(2)},
      /* int64_t O_strides[3] = */ {o.strides(0), o.strides(1), o.strides(2)}};

  dim3 grid_dim(params.H, params.qL, params.B);
  dim3 block_dim(1024, 1, 1);

  dispatch_float_types(o.dtype(), "kernel_sdpav_1pass", [&](auto type_tag) {
    dispatch_bool(do_causal_, [&](auto do_causal) {
      dispatch_headdim(params.D, [&](auto headdim) {
        using DataType = cuda_type_t<MLX_GET_TYPE(type_tag)>;

        auto kernel = cu::kernel_sdpav_1pass<DataType, do_causal(), headdim()>;
        encoder.add_kernel_node(
            kernel,
            grid_dim,
            block_dim,
            0,
            q.data<DataType>(),
            k.data<DataType>(),
            v.data<DataType>(),
            o.data<DataType>(),
            params);
      });
    });
  });
}

void sdpa_vector_2pass_fallback(
    const Stream& s,
    cu::CommandEncoder& encoder,
    const array& q,
    const array& k,
    const array& v,
    const float scale,
    array& o,
    bool do_causal_ = false) {
  cu::AttnParams params{
      /* int B = */ q.shape(0),
      /* int H = */ q.shape(1),
      /* int D = */ q.shape(3),

      /* int qL = */ q.shape(2),
      /* int kL = */ k.shape(2),

      /* int gqa_factor = */ q.shape(1) / k.shape(1),
      /* float scale = */ scale,

      /* int64_t Q_strides[3] = */ {q.strides(0), q.strides(1), q.strides(2)},
      /* int64_t K_strides[3] = */ {k.strides(0), k.strides(1), k.strides(2)},
      /* int64_t V_strides[3] = */ {v.strides(0), v.strides(1), v.strides(2)},
      /* int64_t O_strides[3] = */ {o.strides(0), o.strides(1), o.strides(2)}};

  // Allocate the intermediates
  int blocks = 32;

  Shape intermediate_shape;
  intermediate_shape.reserve(o.ndim() + 1);
  intermediate_shape.insert(
      intermediate_shape.end(), o.shape().begin(), o.shape().end() - 1);
  intermediate_shape.push_back(blocks);
  intermediate_shape.push_back(o.shape().back());

  array intermediate(intermediate_shape, float32, nullptr, {});
  intermediate_shape.pop_back();
  array sums(intermediate_shape, float32, nullptr, {});
  array maxs(std::move(intermediate_shape), float32, nullptr, {});

  intermediate.set_data(allocator::malloc(intermediate.nbytes()));
  sums.set_data(allocator::malloc(sums.nbytes()));
  maxs.set_data(allocator::malloc(maxs.nbytes()));

  encoder.add_temporary(intermediate);
  encoder.add_temporary(sums);
  encoder.add_temporary(maxs);

  dispatch_float_types(o.dtype(), "kernel_sdpav_2pass", [&](auto type_tag) {
    dispatch_bool(do_causal_, [&](auto do_causal) {
      dispatch_headdim(params.D, [&](auto headdim) {
        using DataType = cuda_type_t<MLX_GET_TYPE(type_tag)>;

        {
          auto kernel =
              cu::kernel_sdpav_2pass_1<DataType, do_causal(), headdim()>;

          encoder.set_input_array(q);
          encoder.set_input_array(k);
          encoder.set_input_array(v);
          encoder.set_output_array(intermediate);
          encoder.set_output_array(sums);
          encoder.set_output_array(maxs);

          dim3 grid_dim(params.H, params.qL, params.B * 32);
          dim3 block_dim(8 * 32, 1, 1);

          encoder.add_kernel_node(
              kernel,
              grid_dim,
              block_dim,
              0,
              q.data<DataType>(),
              k.data<DataType>(),
              v.data<DataType>(),
              intermediate.data<float>(),
              sums.data<float>(),
              maxs.data<float>(),
              params);
        }

        {
          auto kernel =
              cu::kernel_sdpav_2pass_2<DataType, do_causal(), headdim()>;

          encoder.set_input_array(intermediate);
          encoder.set_input_array(sums);
          encoder.set_input_array(maxs);
          encoder.set_output_array(o);

          dim3 grid_dim(params.H, params.qL, params.B);
          dim3 block_dim(1024, 1, 1);

          encoder.add_kernel_node(
              kernel,
              grid_dim,
              block_dim,
              0,
              intermediate.data<float>(),
              sums.data<float>(),
              maxs.data<float>(),
              o.data<DataType>(),
              params);
        }
      });
    });
  });
}

void sdpa_vector_fallback(
    const Stream& s,
    cu::CommandEncoder& encoder,
    const array& q,
    const array& k,
    const array& v,
    const float scale,
    array& o,
    bool do_causal_ = false) {
  int kL = k.shape(2);

  if (kL > 1024) {
    return sdpa_vector_2pass_fallback(
        s, encoder, q, k, v, scale, o, do_causal_);
  } else {
    return sdpa_vector_1pass_fallback(
        s, encoder, q, k, v, scale, o, do_causal_);
  }
}

struct SDPACacheKey {
  int device_id;
  fe::DataType_t cudnn_type;

  int B;
  int H;
  int D;

  int qL;
  int kL;

  int gqa_factor;
  float scale;

  int64_t Q_strides[3];
  int64_t K_strides[3];
  int64_t V_strides[3];
  int64_t O_strides[3];

  bool generate_stats;
  bool causal_mask;
};

auto& sdpa_cache() {
  static LRUBytesKeyCache<SDPACacheKey, std::shared_ptr<fe::graph::Graph>>
      cache(
          /* capacity */ 128);
  return cache;
}

#define Q_UID 1
#define K_UID 2
#define V_UID 3
#define O_UID 4
#define STATS_UID 5

std::shared_ptr<fe::graph::Graph> get_sdpa_forward_graph(
    cu::CommandEncoder& encoder,
    const SDPACacheKey& cache_key) {
  // Check if graph has already been fully built
  if (auto it = sdpa_cache().find(cache_key); it != sdpa_cache().end()) {
    return it->second;
  }

  // Set up new graph
  auto graph = std::make_shared<fe::graph::Graph>();

  graph->set_io_data_type(cache_key.cudnn_type)
      .set_intermediate_data_type(fe::DataType_t::FLOAT)
      .set_compute_data_type(fe::DataType_t::FLOAT);

  auto Q = graph->tensor(
      fe::graph::Tensor_attributes()
          .set_name("Q")
          .set_uid(Q_UID)
          .set_dim({cache_key.B, cache_key.H, cache_key.qL, cache_key.D})
          .set_stride(
              {cache_key.Q_strides[0],
               cache_key.Q_strides[1],
               cache_key.Q_strides[2],
               1}));

  int h_kv = cache_key.H / cache_key.gqa_factor;
  auto K =
      graph->tensor(fe::graph::Tensor_attributes()
                        .set_name("K")
                        .set_uid(K_UID)
                        .set_dim({cache_key.B, h_kv, cache_key.kL, cache_key.D})
                        .set_stride(
                            {cache_key.K_strides[0],
                             cache_key.K_strides[1],
                             cache_key.V_strides[2],
                             1}));

  auto V =
      graph->tensor(fe::graph::Tensor_attributes()
                        .set_name("V")
                        .set_uid(V_UID)
                        .set_dim({cache_key.B, h_kv, cache_key.kL, cache_key.D})
                        .set_stride(
                            {cache_key.V_strides[0],
                             cache_key.V_strides[1],
                             cache_key.V_strides[2],
                             1}));

  auto sdpa_options = fe::graph::SDPA_attributes()
                          .set_name("flash_attention")
                          .set_is_inference(!cache_key.generate_stats)
                          .set_attn_scale(cache_key.scale);

  if (cache_key.causal_mask && cache_key.qL > 1) {
    sdpa_options.set_diagonal_alignment(fe::DiagonalAlignment_t::TOP_LEFT)
        .set_diagonal_band_right_bound(0);
  }

  auto [O, Stats] = graph->sdpa(Q, K, V, sdpa_options);

  O->set_output(true)
      .set_uid(O_UID)
      .set_dim({cache_key.B, cache_key.H, cache_key.qL, cache_key.D})
      .set_stride(
          {cache_key.O_strides[0],
           cache_key.O_strides[1],
           cache_key.O_strides[2],
           1});

  if (cache_key.generate_stats) {
    Stats->set_output(true)
        .set_data_type(fe::DataType_t::FLOAT)
        .set_uid(STATS_UID);
  }

  // Build and Validate cudnn graph

  auto handle = encoder.device().cudnn_handle();

  // cuDNN only supports native CUDA graphs for sdpa in 9.6 or above.
  if (hipdnnGetVersion() < 90600) {
    auto build_status = graph->build(handle, {fe::HeurMode_t::A});
    if (!build_status.is_good()) {
      throw std::runtime_error(
          "Unable to build cudnn graph for attention."
          " Failed with message: " +
          build_status.get_message());
    }

  } else {
    auto val_status = graph->validate();
    auto op_status = graph->build_operation_graph(handle);

    auto plan_stauts =
        graph->create_execution_plans({cudnn_frontend::HeurMode_t::A});
    if (!plan_stauts.is_good()) {
      throw std::runtime_error(
          "Unable to create exec plan for cudnn attention."
          " Failed with message: " +
          plan_stauts.get_message());
    }

    graph->select_behavior_notes(
        {cudnn_frontend::BehaviorNote_t::SUPPORTS_CUDA_GRAPH_NATIVE_API});

    auto support_status = graph->check_support(handle);
    if (!support_status.is_good()) {
      throw std::runtime_error(
          "No cuda graph support for cudnn attention."
          " Failed with message: " +
          support_status.get_message());
    }

    auto build_status = graph->build_plans(handle);
    if (!build_status.is_good()) {
      throw std::runtime_error(
          "Unable to build cudnn graph for attention."
          " Failed with message: " +
          build_status.get_message());
    }
  }

  auto [it, _] = sdpa_cache().emplace(cache_key, graph);

  return it->second;
}

inline fe::DataType_t dtype_to_cudnn_type(Dtype dtype) {
  switch (dtype) {
    case int8:
      return fe::DataType_t::INT8;
    case int32:
      return fe::DataType_t::INT32;
    case uint8:
      return fe::DataType_t::UINT8;
    case float16:
      return fe::DataType_t::HALF;
    case bfloat16:
      return fe::DataType_t::BFLOAT16;
    case float32:
      return fe::DataType_t::FLOAT;
    case float64:
      return fe::DataType_t::DOUBLE;
    default:
      throw std::runtime_error(fmt::format(
          "Unsupported dtype in SDPA: {}.", dtype_to_string(dtype)));
  }
}

void sdpa_cudnn(
    const Stream& s,
    cu::CommandEncoder& encoder,
    const array& q,
    const array& k,
    const array& v,
    const float scale,
    array& o,
    bool do_causal_ = false) {
  encoder.set_input_array(q);
  encoder.set_input_array(k);
  encoder.set_input_array(v);
  encoder.set_output_array(o);

  auto cudnn_type = dtype_to_cudnn_type(q.dtype());

  int B = q.shape(0);
  int H = q.shape(1);
  int D = q.shape(3);
  int gqa_factor = q.shape(1) / k.shape(1);

  int qL = q.shape(2);
  int kL = k.shape(2);

  SDPACacheKey cache_key{
      /* int device_id = */ encoder.device().cuda_device(),
      /* fe::DataType_t cudnn_type = */ cudnn_type,

      /* int B = */ B,
      /* int H = */ H,
      /* int D = */ D,

      /* int qL = */ qL,
      /* int kL = */ kL,

      /* int gqa_factor = */ gqa_factor,
      /* float scale = */ scale,

      /* int64_t Q_strides[3] = */ {q.strides(0), q.strides(1), q.strides(2)},
      /* int64_t K_strides[3] = */ {k.strides(0), k.strides(1), k.strides(2)},
      /* int64_t V_strides[3] = */ {v.strides(0), v.strides(1), v.strides(2)},
      /* int64_t O_strides[3] = */ {o.strides(0), o.strides(1), o.strides(2)},

      /* bool generate_stats = */ false,
      /* bool causal_mask = */ do_causal_};

  auto graph = get_sdpa_forward_graph(encoder, cache_key);

  int64_t workspace_size = 0;
  auto workspace_status = graph->get_workspace_size(workspace_size);
  if (!workspace_status.is_good()) {
    throw std::runtime_error("Unable to get workspace for cudnn attention.");
  }

  array workspace(
      allocator::malloc(workspace_size), {int(workspace_size)}, uint8);
  auto workspace_ptr = workspace.data<void>();

  std::unordered_map<int64_t, void*> variant_pack = {
      {Q_UID, const_cast<void*>(q.data<void>())},
      {K_UID, const_cast<void*>(k.data<void>())},
      {V_UID, const_cast<void*>(v.data<void>())},
      {O_UID, o.data<void>()}};

  auto handle = encoder.device().cudnn_handle();
  hipdnnSetStream(handle, encoder.stream());

  // cuDNN only supports native CUDA graphs for sdpa in 9.6 or above.
  if (hipdnnGetVersion() < 90600) {
    auto capture = encoder.capture_context();
    auto exec_status = graph->execute(handle, variant_pack, workspace_ptr);

    if (!exec_status.is_good()) {
      capture.discard = true;
      throw std::runtime_error(
          "Unable to execute cudnn attention."
          " Failed with message: " +
          exec_status.get_message());
    }
  } else {
    hipGraph_t cu_graph;
    hipGraphCreate(&cu_graph, 0);

    std::unique_ptr<hipGraph_t, void (*)(hipGraph_t*)> graph_freer(
        &cu_graph, [](hipGraph_t* p) { hipGraphDestroy(*p); });

    auto cu_graph_status = graph->populate_cuda_graph(
        handle, variant_pack, workspace_ptr, cu_graph);

    if (!cu_graph_status.is_good()) {
      throw std::runtime_error(
          "Unable to add cuda graph for cudnn attention."
          " Failed with message: " +
          cu_graph_status.get_message());
    }

    encoder.add_graph_node(cu_graph);
  }

  encoder.add_temporary(workspace);
}

} // namespace

namespace fast {

bool ScaledDotProductAttention::use_fallback(
    const array& q,
    const array& k,
    const array& v,
    bool has_mask,
    bool has_arr_mask,
    bool do_causal,
    Stream s) {
  if (s.device == Device::cpu) {
    return true;
  }

  const int value_head_dim = v.shape(-1);
  const int query_head_dim = q.shape(-1);
  const int query_sequence_length = q.shape(2);
  const int key_sequence_length = k.shape(2);

  const bool sdpa_supported_head_dim = query_head_dim == value_head_dim &&
      (query_head_dim == 64 || query_head_dim == 96 || query_head_dim == 128);

  return has_arr_mask || !sdpa_supported_head_dim;
}

void ScaledDotProductAttention::eval_gpu(
    const std::vector<array>& inputs,
    array& out) {
  nvtx3::scoped_range r("ScaledDotProductAttention::eval_gpu");

  auto& s = stream();
  auto& encoder = cu::get_command_encoder(s);

  auto& q_pre = inputs[0];
  auto& k_pre = inputs[1];
  auto& v_pre = inputs[2];
  auto& o = out;

  std::vector<array> copies;

  // Define some copy functions to ensure the layout of the inputs is as
  // expected.
  copies.reserve(3);
  auto copy_unless = [&copies, &s](
                         auto predicate, const array& arr) -> const array& {
    if (!predicate(arr)) {
      array arr_copy = contiguous_copy_gpu(arr, s);
      copies.push_back(std::move(arr_copy));
      return copies.back();
    } else {
      return arr;
    }
  };

  auto is_matrix_contiguous = [](const array& arr) {
    return arr.strides(-1) == 1;
  };

  // We are in vector mode ie single query
  if (q_pre.shape(2) <= 1) {
    auto q_copy_unless = [](const array& arr) {
      if (arr.flags().row_contiguous) {
        return true;
      }
      auto& strides = arr.strides();
      auto& shape = arr.shape();
      if (shape[0] == 1 || shape[1] == 1) {
        // If either the batch or head dimension is a singleton, the other can
        // be transposed with the sequence dimension
        auto bidx = shape[0] == 1 ? 1 : 0;
        return (strides[3] == 1) && (strides[2] == shape[3] * shape[bidx]) &&
            (strides[bidx] == shape[3]);
      }
      return false;
    };

    auto kv_copy_unless = [](const array& arr) {
      // keys and values should be copied if:
      // - the last dimension is not contiguous
      // - the batch and head dim are not contiguous
      auto& strides = arr.strides();
      auto& shape = arr.shape();
      if (strides.back() != 1) {
        return false;
      }
      if (shape[0] == 1 || shape[1] == 1) {
        return true;
      }
      return (strides[0] == strides[1] * shape[1]);
    };

    const auto& q = copy_unless(q_copy_unless, q_pre);
    const auto& k = copy_unless(kv_copy_unless, k_pre);
    const auto& v = copy_unless(kv_copy_unless, v_pre);

    for (const auto& cp : copies) {
      encoder.add_temporary(cp);
    }

    // Donate the query if possible
    if (q.is_donatable() && q.flags().row_contiguous && q.size() == o.size()) {
      o.copy_shared_buffer(q);
    } else {
      int64_t str_oD = 1;
      int64_t str_oH = o.shape(3);
      int64_t str_oL = o.shape(1) * str_oH;
      int64_t str_oB = o.shape(2) * str_oL;
      size_t data_size = o.shape(0) * str_oB;

      array::Flags flags{
          /* bool contiguous = */ 1,
          /* bool row_contiguous = */ 0,
          /* bool col_contiguous = */ 0,
      };

      o.set_data(
          allocator::malloc(o.nbytes()),
          data_size,
          {str_oB, str_oH, str_oL, str_oD},
          flags);
    }

    return sdpa_vector_fallback(s, encoder, q, k, v, scale_, o, do_causal_);
    // return sdpa_cudnn(s, encoder, q, k, v, scale_, o, do_causal_);
  }

  // Full attention mode
  else {
    const auto& q = copy_unless(is_matrix_contiguous, q_pre);
    const auto& k = copy_unless(is_matrix_contiguous, k_pre);
    const auto& v = copy_unless(is_matrix_contiguous, v_pre);

    for (const auto& cp : copies) {
      encoder.add_temporary(cp);
    }

    int64_t str_oD = 1;
    int64_t str_oH = o.shape(3);
    int64_t str_oL = o.shape(1) * str_oH;
    int64_t str_oB = o.shape(2) * str_oL;
    size_t data_size = o.shape(0) * str_oB;

    array::Flags flags{
        /* bool contiguous = */ 1,
        /* bool row_contiguous = */ 0,
        /* bool col_contiguous = */ 0,
    };

    o.set_data(
        allocator::malloc(o.nbytes()),
        data_size,
        {str_oB, str_oH, str_oL, str_oD},
        flags);

    return sdpa_cudnn(s, encoder, q, k, v, scale_, o, do_causal_);
  }
}

} // namespace fast

} // namespace mlx::core